#include "hip/hip_runtime.h"
// Copyright (c) 2021 NVIDIA Corporation.  All rights reserved. 
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <string>
#include <cstdio>
#include <iomanip>
#include "dcdread.h"
#include<assert.h>
#include <nvtx3/nvToolsExt.h>

using namespace std;
//Note: CUDA error handling code. This is required as many CUDA calls are async in nature
static void HandleError( hipError_t err,
		const char *file,
		int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ),
				file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//Note declaration of GPU function by using keyword __global__
__global__ void pair_gpu(const double* d_x, const double* d_y, const double* d_z,  unsigned long long int *d_g2, int numatm, int nconf, 
		const double xbox, const double ybox, const double zbox,  int d_bin,  unsigned long long int bl);

int main(int argc , char* argv[] )
{
	double xbox,ybox,zbox;
	double* h_x,*h_y,*h_z;
	double* d_x,*d_y,*d_z;
	unsigned long long int *h_g2,*d_g2;
	int nbin;
	int nthreads,device;
	int numatm,nconf,inconf;
	unsigned long long int near2;
	string file;

	///////////////////////////////////////////////////////////////

	inconf = 10;
	nbin=2000;
	file = "../input/alk.traj.dcd";
	device = 0;
	nthreads = 128;
	HANDLE_ERROR (hipSetDevice(device));//pick the device to use
	///////////////////////////////////////
	std::ifstream infile;
	infile.open(file.c_str());
	if(!infile){
		cout<<"file "<<file.c_str()<<" not found\n";
		return 1;
	}
	assert(infile);


	ofstream pairfile,stwo;
	pairfile.open("RDF.dat");
	stwo.open("Pair_entropy.dat");

	/////////////////////////////////////////////////////////
	dcdreadhead(&numatm,&nconf,infile);
	cout<<"Dcd file has "<< numatm << " atoms and " << nconf << " frames"<<endl;
	if (inconf>nconf) cout << "nconf is reset to "<< nconf <<endl;
	else
	{nconf=inconf;}
	cout<<"Calculating RDF for " << nconf << " frames"<<endl;
	////////////////////////////////////////////////////////

	unsigned long long int sizef= nconf*numatm*sizeof(double);
	unsigned long long int sizebin= nbin*sizeof(unsigned long long int);

	//Note:Allocate memory on CPU
	HANDLE_ERROR(hipHostAlloc((void **)&h_x, sizef, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void **)&h_y, sizef, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void **)&h_z, sizef, hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void **)&h_g2, sizebin, hipHostMallocDefault));

	//Todo: Allocate memory on GPU.
	HANDLE_ERROR(hipMalloc(); //For d_x
	HANDLE_ERROR(hipMalloc(); //For d_y
	HANDLE_ERROR(hipMalloc(); //For d_z
	HANDLE_ERROR(hipMalloc(); //For d_g2

	HANDLE_ERROR (hipPeekAtLastError());

	memset(h_g2,0,sizebin);

	/////////reading cordinates//////////////////////////////////////////////
	nvtxRangePush("Read_File");
	double ax[numatm],ay[numatm],az[numatm];
	for (int i=0;i<nconf;i++) {
	dcdreadframe(ax,ay,az,infile,numatm,xbox,ybox,zbox);
	for (int j=0;j<numatm;j++){
	h_x[i*numatm+j]=ax[j];
	h_y[i*numatm+j]=ay[j];
	h_z[i*numatm+j]=az[j];
	}
	}
	nvtxRangePop(); //pop for REading file


	nvtxRangePush("Pair_Calculation");
	//Todo: Copy the data from Host to Device before calculation on GPU
	HANDLE_ERROR(hipMemcpy(dest, source, ,));
	HANDLE_ERROR(hipMemcpy(dest, source, , ));
	HANDLE_ERROR(hipMemcpy(dest, source, , ));
	HANDLE_ERROR(hipMemcpy(dest, source, , ));

	cout<<"Reading of input file and transfer to gpu is completed"<<endl;
	//////////////////////////////////////////////////////////////////////////

	near2=nthreads*(int(0.5*numatm*(numatm-1)/nthreads)+1);
	unsigned long long int nblock = (near2/nthreads);

	cout<<"Initial blocks are "<<nblock<<" "<<", now changing to ";

	int maxblock=65535;
	int bl;
	int blockloop= int(nblock/maxblock);
	if (blockloop != 0) {
		nblock=maxblock;
	}
	cout<<nblock<<" and will run over "<<(blockloop+1)<<" blockloops"<<endl;

	for (bl=0;bl<(blockloop+1);bl++) {
		//cout <<bl<<endl;
		//Todo: Fill the number of blocks and threads and pass the right device pointers
		pair_gpu<<< , >>> (, , , , numatm, nconf, xbox, ybox, zbox, nbin, bl);

		HANDLE_ERROR (hipPeekAtLastError());
		HANDLE_ERROR(hipDeviceSynchronize());
	}

	//Todo: Copy d_ge back from Device to Host
	HANDLE_ERROR(hipMemcpy(dest, source, , ));

	nvtxRangePop(); //Pop for Pair Calculation

	double pi=acos(-1.0l);
	double rho=(numatm)/(xbox*ybox*zbox);
	double norm=(4.0l*pi*rho)/3.0l;
	double rl,ru,nideal;
	double g2[nbin];
	double r,gr,lngr,lngrbond,s2=0.0l,s2bond=0.0l;
	double box=min(xbox,ybox);
	box=min(box,zbox);
	double del=box/(2.0l*nbin);
	nvtxRangePush("Entropy_Calculation");
	for (int i=0;i<nbin;i++) {
		//      cout<<i+1<<" "<<h_g2[i]<<endl;
		rl=(i)*del;
		ru=rl+del;
		nideal=norm*(ru*ru*ru-rl*rl*rl);
		g2[i]=(double)h_g2[i]/((double)nconf*(double)numatm*nideal);
		r=(i)*del;
		pairfile<<(i+0.5l)*del<<" "<<g2[i]<<endl;
		if (r<2.0l) {
			gr=0.0l;
		}
		else {
			gr=g2[i];
		}
		if (gr<1e-5) {
			lngr=0.0l;
		}
		else {
			lngr=log(gr);
		}

		if (g2[i]<1e-6) {
			lngrbond=0.0l;
		}
		else {
			lngrbond=log(g2[i]);
		}
		s2=s2-2.0l*pi*rho*((gr*lngr)-gr+1.0l)*del*r*r;
		s2bond=s2bond-2.0l*pi*rho*((g2[i]*lngrbond)-g2[i]+1.0l)*del*r*r;

	}
	nvtxRangePop(); //Pop for Entropy Calculation
	stwo<<"s2 value is "<<s2<<endl;
	stwo<<"s2bond value is "<<s2bond<<endl;



	//Note: Freeing up the GPU memory
	cout<<"\n\n\n#Freeing Device memory"<<endl;
	HANDLE_ERROR(hipFree(d_x));
	HANDLE_ERROR(hipFree(d_y));
	HANDLE_ERROR(hipFree(d_z));
	HANDLE_ERROR(hipFree(d_g2));

	cout<<"#Freeing Host memory"<<endl;
	HANDLE_ERROR(hipHostFree ( h_x ) );
	HANDLE_ERROR(hipHostFree ( h_y ) );
	HANDLE_ERROR(hipHostFree ( h_z ) );
	HANDLE_ERROR(hipHostFree ( h_g2 ) );

	cout<<"#Number of atoms processed: "<<numatm<<endl<<endl;
	cout<<"#Number of confs processed: "<<nconf<<endl<<endl;
	cout<<"#number of threads used: "<<nthreads<<endl<<endl;
	return 0;
}

//Todo: Convert the call to GPU call by adding right keyword
void pair_gpu(
		const double* d_x, const double* d_y, const double* d_z, 
		unsigned long long int *d_g2, int numatm, int nconf, 
		const double xbox,const double ybox,const double zbox,int d_bin,  unsigned long long int bl)
{
	double r,cut,dx,dy,dz;
	int ig2,id1,id2;
	double box;
	box=min(xbox,ybox);
	box=min(box,zbox);

	double del=box/(2.0*d_bin);
	cut=box*0.5;
	int thisi;
	double n;

	//Todo: Write indexing logic using threads and blocks
	int i =


		int maxi = min(int(0.5*numatm*(numatm-1)-(bl*65535*128)),(65535*128));

	if ( i < maxi ) {
		thisi=bl*65535*128+i;

		n=(0.5)*(1+ ((double) sqrt (1.0+4.0*2.0*thisi)));
		id1=int(n);
		id2=thisi-(0.5*id1*(id1-1));

		for (int frame=0;frame<nconf;frame++){
			dx=d_x[frame*numatm+id1]-d_x[frame*numatm+id2];
			dy=d_y[frame*numatm+id1]-d_y[frame*numatm+id2];
			dz=d_z[frame*numatm+id1]-d_z[frame*numatm+id2];

			dx=dx-xbox*(round(dx/xbox));
			dy=dy-ybox*(round(dy/ybox));
			dz=dz-zbox*(round(dz/zbox));

			r=sqrtf(dx*dx+dy*dy+dz*dz);
			if (r<cut) {
				ig2=(int)(r/del);
				//Note: Usage of atomic function
				atomicAdd(&d_g2[ig2],2) ;
			}
		}
	}
}


