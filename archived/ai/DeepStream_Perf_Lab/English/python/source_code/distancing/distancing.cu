#include "hip/hip_runtime.h"

/* Copyright (c) 2012, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */



#include <pybind11/stl.h>
#include <thread>
#include <vector>
#include <cmath>

using namespace std;
namespace py = pybind11;
typedef tuple<float, float, float> centroid;

float compute_dist(centroid& p1, centroid& p2)
{
    float x1, y1, h1, x2, y2, h2;
    std::tie(x1, y1, h1) = p1;
    std::tie(x2, y2, h2) = p2;
    float dx = x2 - x1;
    float dy = y2 - y1;

    float lx = dx * 170 * (1/h1 + 1/h2) / 2;
    float ly = dy * 170 * (1/h1 + 1/h2) / 2;

    float l = sqrt(lx*lx + ly*ly);
    return l;
}

float compute_min_dist(int p, centroid& point, vector<centroid>& points) 
{
    vector<float> distances;
    for (auto & p2 : points) {
        distances.push_back(compute_dist(point, p2));
    }
    distances[p] = 1000000.0;
    float min_dist = *std::min_element(distances.begin(), distances.end());
    return min_dist;
}

vector<float> get_min_distances(vector<centroid>& points)
{
    vector<float> out;
    for (int p = 0; p < points.size(); p++) {
        float min_dist = compute_min_dist(p, points[p], points);
        out.push_back(min_dist);
    }
    return out;
}

PYBIND11_MODULE(distancing, m) {
    m.def("get_min_distances", &get_min_distances, "Get min distances");
}
